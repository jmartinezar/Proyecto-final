#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void matrixMul(const double *A, const double *B, double *C, int width_A, int high_A, int width_B, int high_B)
{
	if(width_A != high_B)
	{
		std::cout << "The size of matrices isn't correct\n";
		return;
	}

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < high_A && col < width_B)
	{
		double value = 0;
		for (int k = 0; k < width_A; ++k)
		{
			value += A[row * width_A + k] * B[k * width_B + col];
        	}
		C[row * width_B + col] = value;
	}
}

int main(void)
{
    int width_A = 1000;
    int high_A = 1000;
    int width_B = 1000;
    int high_B = 1000;
    
    size_t size_A = width_A * high_A * sizeof(double);
    size_t size_B = width_B * high_B * sizeof(double);
    size_t size_C = width_B * high_A * sizeof(double);

    double *h_A = (double *)malloc(size_A);
    double *h_B = (double *)malloc(size_B);
    double *h_C = (double *)malloc(size_C);

    for (int i = 0; i < width_A * high_A; i++)
    {
        h_A[i] = 1.0;
    }

    for (int i = 0; i < width_B * high_B; i++)
    {
        h_B[i] = 2.0;
    }

    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size_A);
    hipMalloc((void **)&d_B, size_B);
    hipMalloc((void **)&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    dim3 dimGrid((width_B + dimBlock.x - 1) / dimBlock.x, (high_A + dimBlock.y - 1) / dimBlock.y);

    matrixMul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width);

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++)
    {
        printf("C[%d] = %f\n", i*width_B+1, h_C[i*width_B+1]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

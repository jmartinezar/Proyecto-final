#include <stdio.h>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

__global__ void matrixMul(const double *A, const double *B, double *C, int width_A, int high_A, int width_B, int high_B)
{
	if(width_A != high_B)
	{
		printf("The size of matrices isn't correct\n");
		return;
	}

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < high_A && col < width_B)
	{
		double value = 0;
		for (int k = 0; k < width_A; ++k)
		{
			value += A[row * width_A + k] * B[k * width_B + col];
        	}
		C[row * width_B + col] = value;
	}
}

int main(int argc, char *argv[])
{  
    if (argc < 3) {
      std::cerr << "Uso de: " << argv[0] << " <matrix_size> <number_of_threads>" << std::endl;
      return 1;
    }

    int size = std::atoi(argv[1]);
    int numberOfThreads = std::atoi(argv[2]);
    
    // TODO: rename high -> height
    int width_A = size;
    int high_A = size;
    int width_B = size;
    int high_B = size;

    size_t size_A = width_A * high_A * sizeof(double);
    size_t size_B = width_B * high_B * sizeof(double);
    size_t size_C = width_B * high_A * sizeof(double);

    double *h_A = (double *)malloc(size_A);
    double *h_B = (double *)malloc(size_B);
    double *h_C = (double *)malloc(size_C);

    for (int i = 0; i < width_A * high_A; i++)
    {
        h_A[i] = 1.0;
    }

    for (int i = 0; i < width_B * high_B; i++)
    {
        h_B[i] = 2.0;
    }

    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size_A);
    hipMalloc((void **)&d_B, size_B);
    hipMalloc((void **)&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    dim3 dimBlock(numberOfThreads, numberOfThreads);
    dim3 dimGrid((width_B + dimBlock.x - 1) / dimBlock.x, (high_A + dimBlock.y - 1) / dimBlock.y);

    auto start = std::chrono::system_clock::now(); //start time
    matrixMul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width_A, high_A, width_B, high_B);
    hipDeviceSynchronize();
    auto end = std::chrono::system_clock::now(); //end time

    std::chrono::duration<double> elapsed_seconds = end-start;

    // Total time
    double wtime = elapsed_seconds.count();

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++)
    {
        fprintf(stderr,"C[%d] = %f\n", i*width_B+1, h_C[i*width_B+1]);
    }

    // Prints size and elapsed time in matrix multiplication
    std::cout << size << "\t" << wtime << std::endl;
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
